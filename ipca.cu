#include "hip/hip_runtime.h"
#include <stdio.h>
#include <jpeglib.h>
#include <hip/hip_runtime.h>

#define COUNT 300
#define DIMENSION 16
#define STRIDE 3072
#define AMNESIC 2.0
#define SIZE 32

double* loadJpeg(char* path, double* out=NULL);
bool saveJpeg(char* path, double* i, int height, int width, double min, double max);
unsigned char stepInt(double v, double min = -1.0, double max = 1.0);
double searchMin(double* img, int height, int width);
double searchMax(double* img, int height, int width);
double average(double* img, int height, int width);

__global__ void kernel(int* tableFrame)
{
        int tid = blockIdx.x * blockDim.x + threadIdx.x;
        tableFrame[tid] = tid;
}

__global__ void ipca_kernel( int current, int length, double* tableIn, double* tableU, double* tableV, int* tableFrame ) 
{
	double* imgA = (double*)malloc(sizeof(double)*STRIDE);
	double* imgB = (double*)malloc(sizeof(double)*STRIDE);
	double* imgC = (double*)malloc(sizeof(double)*STRIDE);

	///// thread id -> dimension id
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	tableFrame[tid] = -1;

	for(int f = -tid; f < length; f++)
	{
		__syncthreads();

		int frame = current + f;

		double* strideIn = tableIn + STRIDE * f;
		double* strideV = tableV + STRIDE * tid;
		double* strideU = tableU + STRIDE * tid;

		if(tid == 0)
		{
			for(int s=0; s<STRIDE; s++) strideU[s] = strideIn[s];
//for(int s=0; s<STRIDE; s++) if(strideIn[s]==0) strideU[0]+=1.0;
		}

		if(tid == frame)
		{
			for(int s=0; s<STRIDE; s++) strideV[s] = strideU[s];
			tableFrame[tid] = f;
			continue;
		}
		if(tid > frame) continue;

		///// Vi(n) = [a= (n-1-l)/n * Vi(n-1)] + [b= (1+l)/n * Ui(n)T Vi(n-1)/|Vi(n-1)| * Ui(n) ]
		double nrmV = 0;
		for(int s=0; s<STRIDE; s++){ nrmV += strideV[s]*strideV[s]; }
		nrmV = sqrt(nrmV);
		double dotUV = 0;
		for(int s=0; s<STRIDE; s++){ dotUV += strideV[s]*strideU[s]; }
		double scalerA = ((double)frame - 1.0 - AMNESIC) / frame;
		double scalerB = (1.0 + AMNESIC) * dotUV / ((double)frame * nrmV);

		for(int s=0; s<STRIDE; s++) imgA[s] = strideV[s] * scalerA;
		for(int s=0; s<STRIDE; s++) imgB[s] = strideU[s] * scalerB;
		for(int s=0; s<STRIDE; s++) strideV[s] = imgA[s] + imgB[s];

		///// Ui+1(n) = Ui(n) - [c= Ui(n)T Vi(n)/|Vi(n)| * Vi(n)/|Vi(n)| ]
		if(tid >= DIMENSION - 1) continue;

		nrmV = 0;
		for(int s=0; s<STRIDE; s++){ nrmV += strideV[s]*strideV[s]; }
		nrmV = sqrt(nrmV);
		dotUV = 0;
		for(int s=0; s<STRIDE; s++){ dotUV += strideV[s]*strideU[s]; }
		double scalerC = dotUV / (nrmV * nrmV);

		for(int s=0; s<STRIDE; s++) imgC[s] = strideV[s] * scalerC;
		for(int s=0; s<STRIDE; s++) strideU[STRIDE + s] = strideU[s] - imgC[s];
		
		tableFrame[tid] = f;
	}
	tableFrame[tid] = length;
	free(imgA);
	free(imgB);
	free(imgC);
}

int main(void)
{
	double* images = new double[STRIDE * COUNT];
	double* U = new double[STRIDE * DIMENSION];
	double* V = new double[STRIDE * DIMENSION];
	int* Frame = new int[DIMENSION];

	printf("start count=%d\n", COUNT);
	char path[256];
	for(int c = 0; c < COUNT; c++)
	{
		sprintf(path, "data/%04d.jpg", c);
		loadJpeg(path, images + STRIDE * c);
	}
	for(int p = 0; p < STRIDE * DIMENSION; p++)
	{
		U[p] = 0;
		V[p] = 0;
	}
	double *tableIn, *tableU, *tableV;
	int *tableFrame;
	size_t sizeIn = sizeof(double)*STRIDE*COUNT;
	size_t sizeU = sizeof(double)*STRIDE*DIMENSION;
	size_t sizeV = sizeof(double)*STRIDE*DIMENSION;
	size_t sizeFrame = sizeof(int)*DIMENSION;
 	hipMalloc(&tableIn, sizeIn);
 	hipMalloc(&tableU, sizeU);
 	hipMalloc(&tableV, sizeV);
	hipMalloc(&tableFrame, sizeFrame);
        printf("0. %s\n", hipGetErrorString(hipGetLastError()));

	hipMemcpy(tableIn, images, sizeIn, hipMemcpyHostToDevice);
	hipMemcpy(tableU, U, sizeU, hipMemcpyHostToDevice);
	hipMemcpy(tableV, V, sizeV, hipMemcpyHostToDevice);
        printf("1. %s\n", hipGetErrorString(hipGetLastError()));

	dim3 grid(1,1,1);
	dim3 block(16,1,1);
//	kernel<<<grid, block>>>(tableFrame);
	ipca_kernel<<<grid, block>>>(0, COUNT, tableIn, tableU, tableV, tableFrame);
	printf("2. %s\n", hipGetErrorString(hipGetLastError()));

	hipMemcpy(U, tableU, sizeU, hipMemcpyDeviceToHost);
	hipMemcpy(V, tableV, sizeV, hipMemcpyDeviceToHost);
	hipMemcpy(Frame, tableFrame, sizeFrame, hipMemcpyDeviceToHost);
        printf("3. %s\n", hipGetErrorString(hipGetLastError()));

	for(int d = 0; d < DIMENSION; d++)
	{
		double* img = V + STRIDE * d;
		double min = searchMin(img, SIZE, SIZE);
		double max = searchMax(img, SIZE, SIZE);
		double ave = average(img, SIZE, SIZE);
		printf("%02d: min=%f, max=%f ave=%f frame=%d\n", d, min, max, ave, Frame[d]);
		sprintf(path, "result/%02d.jpg", d);
		saveJpeg(path, img, SIZE, SIZE, min, max);
	}

        hipFree(tableIn);
        hipFree(tableU);
        hipFree(tableV);
        hipFree(tableFrame);

	delete(images);
	delete(U);
	delete(V);
	delete(Frame);
	printf("end\n");
}

double* loadJpeg(char* path, double* out)
{
	struct jpeg_decompress_struct cinfo;
	struct jpeg_error_mgr jerr;

	FILE *infile;

	JSAMPARRAY img;
	int i, j;
	int width;
	int height;

	// initialize JPEG object
	cinfo.err = jpeg_std_error( &jerr );
	jpeg_create_decompress( &cinfo );

	// open file
	infile = fopen( path, "rb" );
	if(infile == NULL) return NULL;
	jpeg_stdio_src( &cinfo, infile );

	// read header
	jpeg_read_header( &cinfo, TRUE );

	// start decompress
	jpeg_start_decompress( &cinfo );

	// get height, width
	width = cinfo.output_width;
	height = cinfo.output_height;

	// prepare memory
	img = (JSAMPARRAY)malloc( sizeof( JSAMPROW ) * height );
	for ( i = 0; i < height; i++ ) {
		img[i] = (JSAMPROW)calloc( sizeof( JSAMPLE ), 3 * width );
	}

	// retrieve
	while( cinfo.output_scanline < cinfo.output_height ) {
		jpeg_read_scanlines( &cinfo,
			img + cinfo.output_scanline,
			cinfo.output_height - cinfo.output_scanline
		);
	}

	// end decompress
	jpeg_finish_decompress( &cinfo );

	// destroy JPEG object
	jpeg_destroy_decompress( &cinfo );

	// close file
	fclose( infile );

	// to double array
	if(out == NULL) out = new double[width*height*3];
	for ( i = 0; i < height; i++ ){
		for ( j = 0; j < width; j++ ) {
			int loc = (width*i+j)*3;
			out[loc + 0] = (double)img[i][j*3+0] / 255.0;
			out[loc + 1] = (double)img[i][j*3+1] / 255.0;
			out[loc + 2] = (double)img[i][j*3+2] / 255.0;
		}
	}

	// free memory
	for ( i = 0; i < height; i++ ) free( img[i] );
	free( img );

	//printf("jpeg:%s (%d,%d) min=%f max=%f ave=%f\n", path, height, width, searchMin(out, height, width), searchMax(out, height, width), average(out, height, width));
	return out;
}

unsigned char stepInt(double v, double min, double max)
{
	if(v <= min) return 0;
	if(v >= max) return 255;
	return (unsigned char)( (v - min) / (max - min) * 255.0 );
}

bool saveJpeg(char* path, double* data, int height, int width, double min, double max)
{
	/* JPEG Object, Error handling */
	struct jpeg_compress_struct cinfo;
	struct jpeg_error_mgr jerr;

	/* Error handling for default */
	cinfo.err = jpeg_std_error(&jerr);

	/* initiazlie JPEG Object */
	jpeg_create_compress(&cinfo);

	/* open output file */
	FILE *fp = fopen(path, "wb");
	if (fp == NULL) {
		fprintf(stderr, "cannot open %s\n", path);
		return false;
	}
	jpeg_stdio_dest(&cinfo, fp);

	/* image settings */
	cinfo.image_width = width;
	cinfo.image_height = height;
	cinfo.input_components = 3;
	cinfo.in_color_space = JCS_RGB;
	jpeg_set_defaults(&cinfo);
	jpeg_set_quality(&cinfo, 75, TRUE);

	/* start compressing */
	jpeg_start_compress(&cinfo, TRUE);

	/* RGB */
	JSAMPARRAY img = (JSAMPARRAY) malloc(sizeof(JSAMPROW) * height);
	for (int i = 0; i < height; i++) {
		img[i] = (JSAMPROW) malloc(sizeof(JSAMPLE) * 3 * width);
		for (int j = 0; j < width; j++) {
			int loc = (i * width + j) * 3;
			img[i][j*3 + 0] = stepInt( data[loc + 0], min, max);
			img[i][j*3 + 1] = stepInt( data[loc + 1], min, max);
			img[i][j*3 + 2] = stepInt( data[loc + 2], min, max );
		}
	}
	/* write */
	jpeg_write_scanlines(&cinfo, img, height);

	/* end compressing */
	jpeg_finish_compress(&cinfo);

	/* destroy JPEG object */
	jpeg_destroy_compress(&cinfo);

	for (int i = 0; i < height; i++) {
		free(img[i]);
	}
	free(img);
	fclose(fp);
	return true;
}

double searchMin(double* img, int height, int width)
{
	double* pEnd = img + height * width * 3;
	double min = 0;
	for(double* p = img; p < pEnd; p++) if(min > *p) min = *p;
	return min;
}
double searchMax(double* img, int height, int width)
{
	double* pEnd = img + height * width * 3;
	double max = 0;
	for(double* p = img; p < pEnd; p++) if(max < *p) max = *p;
	return max;
}
double average(double* img, int height, int width)
{
	double* pEnd = img + height * width * 3;
	double amount = 0;
	for(double* p = img; p < pEnd; p++) amount += *p;
	return amount / (height * width * 3);
}
